// for google colab
// %%writefile naive_convolution.cu
// !nvcc -arch=sm_75 -o  naive_convolution naive_convolution.cu


#include <iostream>
#include <hip/hip_runtime.h>




#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void direct_convolution(
    const float* input, const float* kernel, float* output,
    int in_channels, int in_h, int in_w,
    int out_channels, int kernel_size, int stride, int pad
) {
    const int out_h = (in_h + 2 * pad - kernel_size) / stride + 1;
    const int out_w = (in_w + 2 * pad - kernel_size) / stride + 1;

    // Output coordinates (x, y) and output channel
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    const int c_out = blockIdx.z;

    if (idx >= out_w || idy >= out_h || c_out >= out_channels) return;

    float sum = 0.0f;

    // Iterate over input channels
    for (int c_in = 0; c_in < in_channels; c_in++) {
        // Iterate over kernel elements
        for (int ky = 0; ky < kernel_size; ky++) {
            for (int kx = 0; kx < kernel_size; kx++) {
                // Input coordinates (adjusted for padding)
                const int in_y = idy * stride + ky - pad;
                const int in_x = idx * stride + kx - pad;

                if (in_y >= 0 && in_y < in_h && in_x >= 0 && in_x < in_w) {
                    const float input_val = input[(c_in * in_h + in_y) * in_w + in_x];
                    const float kernel_val = kernel[((c_out * in_channels + c_in) * kernel_size + ky) * kernel_size + kx];
                    sum += input_val * kernel_val;
                }
            }
        }
    }

    // Write output (add batch dimension)
    output[(c_out * out_h + idy) * out_w + idx] = sum;
}

int main() {
    // Configuration (example: 1024x1024 input, 3x3 kernel)
    const int batch = 1;
    const int in_channels = 3;
    const int in_h = 1024, in_w = 1024;
    const int out_channels = 64;
    const int kernel_size = 3;
    const int stride = 1;
    const int pad = 1;

    const int out_h = (in_h + 2 * pad - kernel_size) / stride + 1;
    const int out_w = (in_w + 2 * pad - kernel_size) / stride + 1;

    float *h_input = new float[batch * in_channels * in_h * in_w];
    float *h_kernel = new float[out_channels * in_channels * kernel_size * kernel_size];
    float *h_output = new float[batch * out_channels * out_h * out_w];

    std::fill(h_input, h_input + batch * in_channels * in_h * in_w, 1.0f);
    std::fill(h_kernel, h_kernel + out_channels * in_channels * kernel_size * kernel_size, 1.0f);

    float *d_input, *d_kernel, *d_output;
    hipMalloc(&d_input, batch * in_channels * in_h * in_w * sizeof(float));
    cudaCheckErrors("hipMalloc d_input failed");
    hipMalloc(&d_kernel, out_channels * in_channels * kernel_size * kernel_size * sizeof(float));
    cudaCheckErrors("hipMalloc d_kernel failed");
    hipMalloc(&d_output, batch * out_channels * out_h * out_w * sizeof(float));
    cudaCheckErrors("hipMalloc d_output failed");

    hipMemcpy(d_input, h_input, batch * in_channels * in_h * in_w * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D d_input failed");
    hipMemcpy(d_kernel, h_kernel, out_channels * in_channels * kernel_size * kernel_size * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D d_kernel failed");

    dim3 block(16, 16);
    dim3 grid(
        (out_w + block.x - 1) / block.x,
        (out_h + block.y - 1) / block.y,
        out_channels
    );

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    cudaCheckErrors("hipEventCreate failed");

    direct_convolution<<<grid, block>>>(d_input, d_kernel, d_output, in_channels, in_h, in_w, out_channels, kernel_size, stride, pad);
    cudaCheckErrors("Warm-up kernel failed");

    hipEventRecord(start);
    cudaCheckErrors("hipEventRecord start failed");
    for (int i = 0; i < 10; ++i) { 
     // Running multiple times for stable measurement
        direct_convolution<<<grid, block>>>(d_input, d_kernel, d_output, in_channels, in_h, in_w, out_channels, kernel_size, stride, pad);
        cudaCheckErrors("Main convolution kernel failed");
    }
    hipEventRecord(stop);
    cudaCheckErrors("hipEventRecord stop failed");
    hipEventSynchronize(stop);
    cudaCheckErrors("hipEventSynchronize failed");

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cudaCheckErrors("hipEventElapsedTime failed");
    std::cout << "Time per convolution: " << milliseconds / 10 << " ms\n";

    hipMemcpy(h_output, d_output, batch * out_channels * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("hipMemcpy D2H failed");

    const float expected_value = in_channels * 4;
    std::cout << "First output value: " << h_output[0] << " (Expected: " << expected_value << ")\n";

    // Cleanup
    delete[] h_input;
    delete[] h_kernel;
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}